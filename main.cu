
#include <hip/hip_runtime.h>
#include <unistd.h>

#include <hip/hip_complex.h>
#include <filesystem>
#include <iostream>

namespace fs = std::filesystem;

__global__ void render_image_cuda_d(uint8_t *img, double x_min, double y_min, double delta_x, double delta_y, int res) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int ind = index; ind < res*res; ind += stride) {

    // calculate the pixels of the image
    int i = ind % res;
    int j = ind / res;
    hipDoubleComplex c = make_hipDoubleComplex(x_min + delta_x * (double)(i) / (double)(res), y_min + delta_y * (double)(j) / (double)(res));

    if (hipCabs(c) > 2) {
        img[ind] = 0;
        return;
    }

    hipDoubleComplex z = c;

    uint8_t k_write = 255;
    for (int k = 1; k < 255 * 100; k++) {
        z = hipCadd(hipCmul(z, z), c);

        if (hipCabs(z) > 2) {
            k_write = k % 255;
            break;
        }
    }

    img[ind] = k_write;
    }

    // return
    return;
}

int main(int argc, char *argv[]) {
    // parameters for the image to render
    double x_min = -2.0;
    double y_min = -2.0;
    double delta_x = 4.0;
    double delta_y = 4.0;
    int res = 256;
    int nblocks = 256;
    int nthreads = 256;

    // create folder
    char dir[50] = "zoom/";
    fs::create_directory(dir);
    // calculate the zoomed images

    // empty grayscale image
    uint8_t *img;
    size_t size = res * res * sizeof(uint8_t);
    hipMallocManaged(&img, size);

    // render the image
    render_image_cuda_d<<<nblocks, nthreads>>>(img, x_min, y_min, delta_x, delta_y, res);

    hipDeviceSynchronize();

    printf("Calculation finished!\n");

    // save the image
    char filename[100];
    sprintf(filename, "%scuda.dat", dir);
    
    FILE *file = fopen(filename, "w");
    for (int i=0; i<res; i++) {
        for (int j=0; j<res; j++) {
            fprintf(file, "%d ", img[i + res*j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);

    // return
    return 0;
}